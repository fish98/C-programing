#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "matrix.h"

__global__ void AddMatrix(double * data1, double * data2, double * data3, int row, int col){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N)
        *(data3+row*ix+iy) =  *(data2+row*ix+iy) + *(data2+row*ix+iy);
}

__global__ void FunctionMatrix(double * data1, double * data3, int row, int col){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N)
        *(data3+row*ix+iy) =  pow(*(data1+row*ix+iy), 2);
}

__host__ void CUDA_AddMatrix(Matrix * M1, Matrix * M2, Matrix * M3){

    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M2->row;
        
    int area1 = M1->col * M1->row;
    int area2 = M2->col * M2->row;

    M3->col = M1->col;
    M3->row = M2->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data2;
    double * GPU_data3;

    int row = M1->row;
    int col = M1->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area1);
    hipMalloc((void**)&GPU_data2, sizeof(double) * area2);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area1, hipMemcpyHostToDevice);
    hipMemcpy(GPU_data2, M2->data, sizeof(double) * area2, hipMemcpyHostToDevice);

    AddMatrix<<<blocks, threads>>>(GPU_data1, GPU_data2, GPU_data3, row, col);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Add Operation is: \n");

    PrintMatrix(M3);
    
    free(M3->data);
}





















__global__ void MultiplyMatrix(double * data1, double * data2, double * data3, int row1, int col1, int row2, int col2){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N){
        int i, j, k;
        for(i = 0; i< col1; i++){
            for(j = 0; j< row2;j++){
                int result = 0;
                for(k = 0; k < row1; k++){
                    result += *(data1+i*row1+k) * *(data2+k*row2+j);
                }
                *(data3+i*row2+j) = result;
            }
        }
    }
}

__host__ void CUDA_MultiplyMatrix(Matrix * M1, Matrix * M2, Matrix * M3){
    
    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M2->row;
        
    int area1 = M1->col * M1->row;
    int area2 = M2->col * M2->row;

    M3->col = M1->col;
    M3->row = M2->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data2;
    double * GPU_data3;

    int row1 = M1->row;
    int col1 = M1->col;

    int row2 = M2->row;
    int col2 = M2->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area1);
    hipMalloc((void**)&GPU_data2, sizeof(double) * area2);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area1, hipMemcpyHostToDevice);
    hipMemcpy(GPU_data2, M2->data, sizeof(double) * area2, hipMemcpyHostToDevice);

    MultiplyMatrix<<<blocks, threads>>>(GPU_data1, GPU_data2, GPU_data3, row1, col1, row2, col2);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Multiply Operation is: \n");

    PrintMatrix(M3);
    
    free(M3->data);
}





















__host__ void CUDA_FunctionMatrix(Matrix * M1, Matrix * M3){

    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M1->row;

    M3->col = M1->col;
    M3->row = M1->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data3;

    int row = M1->row;
    int col = M1->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area, hipMemcpyHostToDevice);

    FunctionMatrix<<<blocks, threads>>>(GPU_data1, GPU_data3, row, col);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Function Operation is: \n");

    PrintMatrix(M3);

    free(M3->data);
}


















// __global__ void Test(double * data){
//     printf("%lf\n", *data);
// }

void PrintMatrix(Matrix * M){
    int i, j;
    for (i = 0; i < M ->col; i++){
        for (j = 0; j < M->row; j++){
            printf("%lf ", *(M->data+i*M->row + j));
        }
        printf("\n");
    }
}
void ReadMatrix(const char * Filename, Matrix * M){

    FILE * fp;

    int area, prefix = 0;

    fp = fopen(Filename, "r");

    fscanf(fp, "%d", &M->row);
    fscanf(fp, "%d", &M->col);

    area = M->col * M->row;

    M->data = (double *)malloc(sizeof(double) * area);

    while(fscanf(fp, "%lf", M->data+prefix++)!= EOF);
}

// __global__ void AddMatrix(Matrix * M1, Matrix * M2, Matrix * M3){
//     int i;

//     // Judge First

//     // if (M1->col != M2->col || M1->row != M2 ->row){
//     //     printf("Invalid Input of Matrix\n");
//     //     printf("Choose Operator '+ or * or f' (enter q for quit)\n");
//     // } else{

//     // Init M3

//     M3->col = M1->col;
//     M3->row = M1->row;
//     M3->data = hipMalloc(sizeof(double) * M3->col * M3->row);

//     for(i = 0; i<M3->col * M3->row; i++){
//         *(M3->data+i) = *(M1->data+i)+*(M2->data+i);
//         }
//     // printf("Output for Add Operation\n");
//     // PrintMatrix(M3);
//     // printf("Choose Operator '+ or * or f' (enter q for quit)\n");
//     // }
// }

// void MiltiplyMatrix(Matrix * M1, Matrix * M2, Matrix * M3){
//     int i, j, k;

//     // Judge First

//     if (M1->row != M2->col){
//         printf("Invalid Input of Matrix\n");
//         printf("Choose Operator (enter q for quit)\n");
//     } else
//     {
//     // Init M3

//     M3->col = M1->col;
//     M3->row = M2->row;
//     M3->data = malloc(sizeof(double) * M3->col * M3->row);
//     }
//     for(i = 0; i< M3->col; i++){
//         for(j = 0; j< M3-> row;j++){
//             int result = 0;
//             for(k = 0; k < M1->row; k++){
//                result += *(M1->data+i*M1->row+k) * *(M2->data+k*M2->row+j);
//             }
//             *(M3->data+i*M3->row+j) = result;
//         }
//     }
//     printf("Output for Miltiply Operation\n");
//     PrintMatrix(M3);
//     printf("Choose Operator '+ or * or f' (enter q for quit)\n"); 
// }

// void FunctionMatrix(Matrix * M1, Matrix * M3){

//     // Init M3

//     int i, power;
//     M3->col = M1->col;
//     M3->row = M1->row;
//     M3->data = malloc(sizeof(double) * M3->col * M3->row);
//     for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = *(M1->data+i);
//         }

//     printf("enter the power of the operation element\n");

//     scanf("%d", &power);
//     if(power > 0){
//         for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = pow(*(M3->data+i), power);
//         }
//     }
//     else if(power == 0){
//          for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = 1;
//         }
//     }
//     else {
//         power = -power;
//         for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = 1 / pow(*(M3->data+i), power);
//         }
//     }
//     printf("Output for Miltiply Operation\n");
//     PrintMatrix(M3);
//     printf("Choose Operator '+ or * or f' (enter q for quit)\n"); 

//     void FunctionMatrix(Matrix * M1, Matrix * M3){

//     // Init M3

//     int i, power;
//     M3->col = M1->col;
//     M3->row = M1->row;
//     M3->data = malloc(sizeof(double) * M3->col * M3->row);
//     for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = *(M1->data+i);
//         }
    
//     power = 2;

//     if(power > 0){
//         for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = pow(*(M3->data+i), power);
//         }
//     }
//     else if(power == 0){
//          for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = 1;
//         }
//     }
//     else {
//         power = -power;
//         for(i = 0; i < M3->col * M3->row; i++){
//             *(M3->data+i) = 1 / pow(*(M3->data+i), power);
//         }
//     }
// }

void TransposeMatrix(double * data, int row, int col){

    int area = col * row;
    int i, j;

    double * TmpData = (double*)malloc(sizeof(double) * area);

    for(i=0;i<row;i++){
        for(j=0;j<col;j++){
            *(TmpData+col*i+j) = *(data+row*j+i);
        }
    }

   for(i=0;i<row;i++){
       for(j=0;j<col;j++){
           *(data+col*i+j) = *(TmpData+col*i+j);
       }
   }
    free(TmpData);
}
