#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "matrix.h"

__global__ void AddMatrix(double * data1, double * data2, double * data3, int row, int col){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N)
        *(data3+row*ix+iy) =  *(data2+row*ix+iy) + *(data2+row*ix+iy);
}

__global__ void FunctionMatrix(double * data1, double * data3, int row, int col){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N)
        *(data3+row*ix+iy) =  pow(*(data1+row*ix+iy), 2);
}

__host__ void CUDA_AddMatrix(Matrix * M1, Matrix * M2, Matrix * M3){

    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M2->row;
        
    int area1 = M1->col * M1->row;
    int area2 = M2->col * M2->row;

    M3->col = M1->col;
    M3->row = M2->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data2;
    double * GPU_data3;

    int row = M1->row;
    int col = M1->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area1);
    hipMalloc((void**)&GPU_data2, sizeof(double) * area2);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area1, hipMemcpyHostToDevice);
    hipMemcpy(GPU_data2, M2->data, sizeof(double) * area2, hipMemcpyHostToDevice);

    AddMatrix<<<blocks, threads>>>(GPU_data1, GPU_data2, GPU_data3, row, col);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Add Operation is: \n");

    PrintMatrix(M3);
    
    free(M3->data);
}

__global__ void MultiplyMatrix(double * data1, double * data2, double * data3, int row1, int col1, int row2, int col2){
    int ix = blockIdx.x*4 + threadIdx.x, iy = blockIdx.y*4 + threadIdx.y;
    if (ix < N && iy < N){
        int i, j, k;
        for(i = 0; i< col1; i++){
            for(j = 0; j< row2;j++){
                int result = 0;
                for(k = 0; k < row1; k++){
                    result += *(data1+i*row1+k) * *(data2+k*row2+j);
                }
                *(data3+i*row2+j) = result;
            }
        }
    }
}

__host__ void CUDA_MultiplyMatrix(Matrix * M1, Matrix * M2, Matrix * M3){
    
    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M2->row;
        
    int area1 = M1->col * M1->row;
    int area2 = M2->col * M2->row;

    M3->col = M1->col;
    M3->row = M2->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data2;
    double * GPU_data3;

    int row1 = M1->row;
    int col1 = M1->col;

    int row2 = M2->row;
    int col2 = M2->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area1);
    hipMalloc((void**)&GPU_data2, sizeof(double) * area2);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area1, hipMemcpyHostToDevice);
    hipMemcpy(GPU_data2, M2->data, sizeof(double) * area2, hipMemcpyHostToDevice);

    MultiplyMatrix<<<blocks, threads>>>(GPU_data1, GPU_data2, GPU_data3, row1, col1, row2, col2);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Multiply Operation is: \n");

    PrintMatrix(M3);
    
    free(M3->data);
}

__host__ void CUDA_FunctionMatrix(Matrix * M1, Matrix * M3){

    dim3 blocks(N/4+1,N/4+1);
    dim3 threads(4,4);

    int area = M1->col * M1->row;

    M3->col = M1->col;
    M3->row = M1->row;

    M3->data = (double *)malloc(sizeof(double) * area);

    double * GPU_data1;
    double * GPU_data3;

    int row = M1->row;
    int col = M1->col;

    hipMalloc((void**)&GPU_data1, sizeof(double) * area);
    hipMalloc((void**)&GPU_data3, sizeof(double) * area);
    
    hipMemcpy(GPU_data1, M1->data, sizeof(double) * area, hipMemcpyHostToDevice);

    FunctionMatrix<<<blocks, threads>>>(GPU_data1, GPU_data3, row, col);

    hipMemcpy(M3->data, GPU_data3, sizeof(double) * area, hipMemcpyDeviceToHost);
    
    hipDeviceReset();
    
    printf("The result of Function Operation is: \n");

    PrintMatrix(M3);

    free(M3->data);
}

void PrintMatrix(Matrix * M){
    int i, j;
    for (i = 0; i < M ->col; i++){
        for (j = 0; j < M->row; j++){
            printf("%lf ", *(M->data+i*M->row + j));
        }
        printf("\n");
    }
}
void ReadMatrix(const char * Filename, Matrix * M){

    FILE * fp;

    int area, prefix = 0;

    fp = fopen(Filename, "r");

    fscanf(fp, "%d", &M->row);
    fscanf(fp, "%d", &M->col);

    area = M->col * M->row;

    M->data = (double *)malloc(sizeof(double) * area);

    while(fscanf(fp, "%lf", M->data+prefix++)!= EOF);
}

void TransposeMatrix(double * data, int row, int col){

    int area = col * row;
    int i, j;

    double * TmpData = (double*)malloc(sizeof(double) * area);

    for(i=0;i<row;i++){
        for(j=0;j<col;j++){
            *(TmpData+col*i+j) = *(data+row*j+i);
        }
    }

   for(i=0;i<row;i++){
       for(j=0;j<col;j++){
           *(data+col*i+j) = *(TmpData+col*i+j);
       }
   }
    free(TmpData);
}
