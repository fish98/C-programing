#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cstdio>
#include "matrix.h"

int main(int argc, char **argv){

    clock_t start, finish;

    Matrix * M1 = (Matrix *)malloc(sizeof(Matrix));
    Matrix * M2 = (Matrix *)malloc(sizeof(Matrix));
    Matrix * M3 = (Matrix *)malloc(sizeof(Matrix));
    
    ReadMatrix("./matrix1", M1);
    ReadMatrix("./matrix2", M2);

    printf("Load Matrix Done\n\n");
    start = clock(); 
    CUDA_AddMatrix(M1, M2, M3);
    finish  = clock();
    printf("Use %lf Seconds To Finish Add Operation \n\n", (double)(finish-start)/1000000);

    start = clock(); 
    CUDA_MultiplyMatrix(M1, M2, M3);
    finish  = clock();
    printf("Use %lf Seconds To Finish Multiply Operation \n\n", (double)(finish-start)/1000000);

    start = clock(); 
    CUDA_FunctionMatrix(M1, M3);
    finish  = clock();
    printf("Use %lf Seconds To Finish MenberWise Operation \n\n", (double)(finish-start)/1000000);

}
